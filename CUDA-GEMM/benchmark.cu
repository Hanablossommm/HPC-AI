#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
int main(){
    unsigned int N=1024;
    unsigned int size=N*N*sizeof(float);
    //initialize data
    float *d_A = (float*)malloc(size);
    float *d_B = (float*)malloc(size);
    float *d_C = (float*)malloc(size);
    // Initialize matrices A and B
    for(unsigned int i=0;i<N*N;i++){
        d_A[i]=1.0f;
        d_B[i]=2.0f;
    }
    float *A,*B,*C;
    //allocate memory on the device
    hipMalloc((void**)&A, size);
    hipMalloc((void**)&B, size);
    hipMalloc((void**)&C, size);
    //copy data to device
    hipMemcpy(A, d_A, size, hipMemcpyHostToDevice);
    hipMemcpy(B, d_B, size, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEventRecord(start);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A, N, B, N, &beta, C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);
    
    hipblasDestroy(handle);
    // copy data back to host
    hipMemcpy(d_C, C, size, hipMemcpyDeviceToHost);
    // Free memory
    free(d_A);
    free(d_B);
    free(d_C);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}