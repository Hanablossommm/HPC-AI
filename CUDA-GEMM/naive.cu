
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void naive_GEMM(float* A,float* B,float*C,unsigned int M,unsigned int N,unsigned int K){
    unsigned int row=blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col=blockIdx.x*blockDim.x+threadIdx.x;
    float sum=0.0f;
    if(row>=M || col>=N) return; // Check bounds
    for(unsigned int i=0;i<K;i++){
        sum+=A[row*K+i]*B[i*N+col];
    }
    C[row*N+col]=sum;
}
int main(){
    unsigned int M=1024;
    unsigned int N=1024;
    unsigned int K=1024;
    unsigned int size=N*N*sizeof(float);
    //initialize data
    float *d_A = (float*)malloc(size);
    float *d_B = (float*)malloc(size);
    float *d_C = (float*)malloc(size);
    // Initialize matrices A and B
    for(unsigned int i=0;i<N*N;i++){
        d_A[i]=1.0f;
        d_B[i]=2.0f;
    }
    float *A,*B,*C;
    //allocate memory on the device
    hipMalloc((void**)&A, size);
    hipMalloc((void**)&B, size);
    hipMalloc((void**)&C, size);
    //copy data to device
    hipMemcpy(A, d_A, size, hipMemcpyHostToDevice);
    hipMemcpy(B, d_B, size, hipMemcpyHostToDevice);
     //cuda events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Free host memory
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, 
             (M + blockDim.y - 1) / blockDim.y);
    hipEventRecord(start);
    naive_GEMM<<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);
    
    /*
      // Check for errors
    cudaError_t err = cudaGetLastError();
    if (err != cudaSuccess) {
        fprintf(stderr, "Error: %s\n", cudaGetErrorString(err));
        return -1; 
    }
    // copy data back to host
    cudaMemcpy(d_C, C, size, cudaMemcpyDeviceToHost);
    // Print a few results
    printf("Result of matrix multiplication (first 10 elements):\n");
    for(unsigned int i=0;i<10;i++){
        printf("%f ", d_C[i]);
    }
     printf("\n");
    */
    // Copy result back to host
    hipMemcpy(d_C, C, size, hipMemcpyDeviceToHost);
    // Free memory
    free(d_A);
    free(d_B);
    free(d_C);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}