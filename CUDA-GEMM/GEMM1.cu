/*improve naive by tiling*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#define TILE_DIM 32
__global__ void mm_kernel(float* A,float* B,float*C,unsigned int N){
    unsigned int row=blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col=blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ float A_s[TILE_DIM][TILE_DIM];
    __shared__ float B_s[TILE_DIM][TILE_DIM];

    float sum=0.0f;
    for(unsigned int i=0;i<N/TILE_DIM;i++){
        A_s[threadIdx.y][threadIdx.x]=A[row*N+i*TILE_DIM+threadIdx.x];
        B_s[threadIdx.y][threadIdx.x]=B[(i*TILE_DIM+threadIdx.y)*N+col];
        __syncthreads();
        for(unsigned int j=0;j<TILE_DIM;j++){
            sum+=A_s[threadIdx.y][j]*B_s[j][threadIdx.x];
        }
        __syncthreads();
    }
    if(row<N && col<N){
        C[row*N+col]=sum;
    }
}
int main(){
    unsigned int N=1024;
    unsigned int size=N*N*sizeof(float);
    //initialize data
    float *d_A = (float*)malloc(size);
    float *d_B = (float*)malloc(size);
    float *d_C = (float*)malloc(size);
    // Initialize matrices A and B
    for(unsigned int i=0;i<N*N;i++){
        d_A[i]=1.0f;
        d_B[i]=2.0f;
    }
    float *A,*B,*C;
    //allocate memory on the device
    hipMalloc((void**)&A, size);
    hipMalloc((void**)&B, size);
    hipMalloc((void**)&C, size);
    //copy data to device
    hipMemcpy(A, d_A, size, hipMemcpyHostToDevice);
    hipMemcpy(B, d_B, size, hipMemcpyHostToDevice);
    //cuda events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Free host memory
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y);
    hipEventRecord(start);
    mm_kernel<<<gridDim, blockDim>>>(A, B, C, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);

    /*
    // Check for errors
    cudaError_t err = cudaGetLastError();
    if (err != cudaSuccess) {
        fprintf(stderr, "Error: %s\n", cudaGetErrorString(err));
        return -1; 
    }
    // copy data back to host
    cudaMemcpy(d_C, C, size, cudaMemcpyDeviceToHost);
    // Print a few results
    printf("Result of matrix multiplication (first 10 elements):\n");
    for(unsigned int i=0;i<10;i++){
        printf("%f ", d_C[i]);
    }
    printf("\n");
    
    */
    hipMemcpy(d_C, C, size, hipMemcpyDeviceToHost);
    // Free memory
    free(d_A);
    free(d_B);
    free(d_C);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}